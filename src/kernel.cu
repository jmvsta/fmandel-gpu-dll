#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"
#include <jni.h>
#include <fstream>
#include <vector>

#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image_write.h"

__global__ void mandelbrot_kernel(unsigned char *image, int width, int height, double x_min, double x_max, double y_min, double y_max,
                  int max_iter, double zoom_x, double zoom_y, int zoom_level) {
    auto px = blockIdx.x * blockDim.x + threadIdx.x;
    auto py = blockIdx.y * blockDim.y + threadIdx.y;

    if (px < width && py < height) {

        double zoomed_x_min = zoom_x - (x_max - x_min) / (2.0f * zoom_level);
        double zoomed_x_max = zoom_x + (x_max - x_min) / (2.0f * zoom_level);
        double zoomed_y_min = zoom_y - (y_max - y_min) / (2.0f * zoom_level);
        double zoomed_y_max = zoom_y + (y_max - y_min) / (2.0f * zoom_level);

        double x0 = zoomed_x_min + (px / (double) width) * (zoomed_x_max - zoomed_x_min);
        double y0 = zoomed_y_min + (py / (double) height) * (zoomed_y_max - zoomed_y_min);
        double x = 0.0;
        double y = 0.0;
        int iteration = 0;

        while (x * x + y * y <= 4.0 && iteration < max_iter) {
            double xtemp = x * x - y * y + x0;
            y = 2 * x * y + y0;
            x = xtemp;
            iteration++;
        }
        auto pixelIndex = (py * width + px) * 3;

        if (iteration == max_iter) {
            image[pixelIndex] = 0;
            image[pixelIndex + 1] = 0;
            image[pixelIndex + 2] = 0;
        } else {
//            image[pixelIndex] = (unsigned char) (255 * iteration / max_iter);
//            image[pixelIndex + 1] = (unsigned char) (255 * iteration / max_iter);
//            image[pixelIndex + 2] = (unsigned char) (255 * iteration / max_iter);
            float hue = 0.6f + static_cast<float>(iteration) / static_cast<float>(max_iter);
            hue = fmod(hue, 1.0f);
            float chroma = 1.0f * 15.0f;
            float h_prime = fmod(hue * 6.0f, 6.0f);
            float xx = chroma * (1.0f - fabs(fmod(h_prime, 2.0f) - 1.0f));

            float r1 = 0, g1 = 0, b1 = 0;

            if (0 <= h_prime && h_prime < 1) {
                r1 = chroma;
                g1 = xx;
                b1 = 0;
            } else if (1 <= h_prime && h_prime < 2) {
                r1 = xx;
                g1 = chroma;
                b1 = 0;
            } else if (2 <= h_prime && h_prime < 3) {
                r1 = 0;
                g1 = chroma;
                b1 = xx;
            } else if (3 <= h_prime && h_prime < 4) {
                r1 = 0;
                g1 = xx;
                b1 = chroma;
            } else if (4 <= h_prime && h_prime < 5) {
                r1 = xx;
                g1 = 0;
                b1 = chroma;
            } else if (5 <= h_prime && h_prime < 6) {
                r1 = chroma;
                g1 = 0;
                b1 = xx;
            }

            float m = 15.0f - chroma;
            image[pixelIndex] = static_cast<unsigned char>((r1 + m) * 255);
            image[pixelIndex + 1] = static_cast<unsigned char>((g1 + m) * 255);
            image[pixelIndex + 2] = static_cast<unsigned char>((b1 + m) * 255);
        }

    }
}

extern "C" {

__declspec(dllexport) unsigned char *allocate_image(int width, int height) {
    unsigned char *image;
    hipMallocManaged(&image, width * height * 3);
    return image;
}

__declspec(dllexport)  void free_image(unsigned char *image) {
    hipFree(image);
}

__declspec(dllexport)  void generate_mandelbrot(unsigned char *image, int width, int height,
                                                double x_center, double y_center,
                                                double x_min, double x_max,
                                                double y_min, double y_max,
                                                int max_iter, int zoom_steps) {
    dim3 threads_per_block(16, 16);
    dim3 num_blocks((width + threads_per_block.x - 1) / threads_per_block.x,
                    (height + threads_per_block.y - 1) / threads_per_block.y);

    mandelbrot_kernel <<<num_blocks, threads_per_block>>>(image, width, height, x_min, x_max, y_min, y_max, max_iter,
                                                          x_center, y_center, zoom_steps);
    hipDeviceSynchronize();
}

__declspec(dllexport)  void save_image(const char *filename, unsigned char *image, int width, int height) {
    std::ofstream ofs(filename, std::ios::binary);
    ofs << "P6\n" << width << " " << height << "\n255\n";
    ofs.write(reinterpret_cast<char *>(image), width * height * 3);
    ofs.close();
}

}

//extern "C" JNIEXPORT jobject JNICALL Java_kt_MandelbrotLibrary_allocateImage(JNIEnv* env, jobject obj, jint width, jint height) {
//    unsigned char* image = allocate_image(width, height);
//    return env->NewDirectByteBuffer(image, width * height * 3);  // Create a direct ByteBuffer from native memory
//}
//
//extern "C" JNIEXPORT void JNICALL Java_kt_MandelbrotLibrary_freeImage(JNIEnv* env, jobject obj, jobject buffer) {
//    unsigned char* image = (unsigned char*)env->GetDirectBufferAddress(buffer);  // Get the pointer from ByteBuffer
//    free_image(image);  // Free the native memory
//}
//
//extern "C" JNIEXPORT void JNICALL Java_kt_MandelbrotLibrary_generateMandelbrot(JNIEnv* env, jobject obj,
//    jobject imageBuffer, jint width, jint height,
//    jdouble xCenter, jdouble yCenter,
//    jdouble xMin, jdouble xMax,
//    jdouble yMin, jdouble yMax,
//    jint maxIter, jint zoomSteps) {
//    // Get the pointer from the ByteBuffer
//    unsigned char* image = (unsigned char*)env->GetDirectBufferAddress(imageBuffer);
//
//    // Check if the pointer is valid
//    if (image == nullptr) {
//        printf("Error: GetDirectBufferAddress returned NULL\n");
//        return;
//    }
//
//    // Call the CUDA function to generate the src set
//    generate_mandelbrot(image, width, height, xCenter, yCenter, xMin, xMax, yMin, yMax, maxIter, zoomSteps);
//}
//
//extern "C" JNIEXPORT void JNICALL Java_kt_MandelbrotLibrary_saveImage(JNIEnv* env, jobject obj,
//    jstring filename, jobject imageBuffer,
//    jint width, jint height) {
//    // Get the pointer from the ByteBuffer
//    unsigned char* image = (unsigned char*)env->GetDirectBufferAddress(imageBuffer);
//
//    // Get the filename from the jstring
//    const char* file = env->GetStringUTFChars(filename, nullptr);
//
//    // Call the CUDA function to save the image
//    save_image(file, image, width, height);
//
//    // Release the filename memory
//    env->ReleaseStringUTFChars(filename, file);
//}
void write_to_memory(void *context, void *data, int size) {
    auto buffer = static_cast<std::vector<unsigned char> *>(context);
    buffer->insert(buffer->end(), (unsigned char *) data, (unsigned char *) data + size);
}


extern "C" JNIEXPORT jbyteArray
JNICALL Java_com_jmvsta_fmandelbackend_MandelbrotLibrary_makePicture(JNIEnv *env, jobject obj,
                                                                     jint width, jint height,
                                                                     jdouble xCenter, jdouble yCenter,
                                                                     jdouble xMin, jdouble xMax,
                                                                     jdouble yMin, jdouble yMax,
                                                                     jint maxIter, jint zoomSteps) {

    unsigned char *image = allocate_image(width, height);

    generate_mandelbrot(image, width, height, xCenter, yCenter, xMin, xMax, yMin, yMax, maxIter, zoomSteps);
    std::vector<unsigned char> jpegData;
    stbi_write_jpg_to_func(write_to_memory, &jpegData, width, height, 3, image, 95);


    free_image(image);
    if (jpegData.size() > static_cast<size_t>(std::numeric_limits<jsize>::max())) {
        return nullptr;
    }

    auto arraySize = static_cast<jsize>(jpegData.size());

    jbyteArray result = env->NewByteArray(arraySize);
    env->SetByteArrayRegion(result, 0, arraySize, reinterpret_cast<jbyte *>(jpegData.data()));
    return result;
}
